hipEvent_t start, stop;

hipEventCreate(&start);
hipEventCreate(&stop);

hipEventRecord(start, 0);

//work kernel

hipEventRecord(stop, 0);
hipEventSynchronize(stop);
float worktime;

hipEventElapsedTime(&worktime, start, stop);

printf("Time = %3.1f ms \n", worktime);
hipEventDestroy(start);
hipEventDestroy(stop);