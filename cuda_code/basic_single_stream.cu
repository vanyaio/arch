#include "hip/hip_runtime.h"
#include <stdio.h>

#define N   (1024*1024)
#define FULL_DATA_SIZE   (N*20)


__global__ void kernel( int *a, int *b, int *c ) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        int idx1 = (idx + 1) % 256;
        int idx2 = (idx + 2) % 256;
        float   as = (a[idx] + a[idx1] + a[idx2]) / 3.0f;
        float   bs = (b[idx] + b[idx1] + b[idx2]) / 3.0f;
        c[idx] = (as + bs) / 2;
    }
}
        

int main( void ) {
    hipDeviceProp_t  prop;
    int whichDevice;
    hipGetDevice( &whichDevice );
    hipGetDeviceProperties( &prop, whichDevice );
    if (!prop.deviceOverlap) {
        printf( "Device will not handle overlaps, so no speed up from streams\n" );
        return 0;
    }

    hipEvent_t     start, stop;
    float           elapsedTime;

    hipStream_t    stream;
    int *host_a, *host_b, *host_c;
    int *dev_a, *dev_b, *dev_c;

    // start the timers
    hipEventCreate( &start );
    hipEventCreate( &stop );

    // initialize the stream
    hipStreamCreate( &stream );

    // allocate the memory on the GPU
    hipMalloc( (void**)&dev_a,
                              N * sizeof(int) );
    hipMalloc( (void**)&dev_b,
                              N * sizeof(int) );
    hipMalloc( (void**)&dev_c,
                              N * sizeof(int) );

    // allocate host locked memory, used to stream
    hipHostAlloc( (void**)&host_a,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault );
    hipHostAlloc( (void**)&host_b,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault );
    hipHostAlloc( (void**)&host_c,
                              FULL_DATA_SIZE * sizeof(int),
                              hipHostMallocDefault );

    for (int i=0; i<FULL_DATA_SIZE; i++) {
        host_a[i] = rand();
        host_b[i] = rand();
    }

    hipEventRecord( start, 0 );
    // now loop over full data, in bite-sized chunks
    for (int i=0; i<FULL_DATA_SIZE; i+= N) {
        // copy the locked memory to the device, async
        hipMemcpyAsync( dev_a, host_a+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream );
        hipMemcpyAsync( dev_b, host_b+i,
                                       N * sizeof(int),
                                       hipMemcpyHostToDevice,
                                       stream );

        kernel<<<N/256,256,0,stream>>>( dev_a, dev_b, dev_c );

        // copy the data from device to locked memory
        hipMemcpyAsync( host_c+i, dev_c,
                                       N * sizeof(int),
                                       hipMemcpyDeviceToHost,
                                       stream );

    }
    // copy result chunk from locked to full buffer
    hipStreamSynchronize( stream );

    hipEventRecord( stop, 0 );

    hipEventSynchronize( stop );
    hipEventElapsedTime( &elapsedTime,
                                        start, stop );
    printf( "Time taken:  %3.1f ms\n", elapsedTime );

    // cleanup the streams and memory
    hipHostFree( host_a );
    hipHostFree( host_b );
    hipHostFree( host_c );
    hipFree( dev_a );
    hipFree( dev_b );
    hipFree( dev_c );
    hipStreamDestroy( stream );

    return 0;
}

