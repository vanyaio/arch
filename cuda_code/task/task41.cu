
#include <hip/hip_runtime.h>
#include "stdio.h"

#define N 4
#define M 4
#define thx 2
#define thy 2

__global__ void add( int *a, int *b, int *c )
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	int j = threadIdx.y + blockIdx.y * blockDim.y;
	int ind = i * N + j;
	c[ind] = a[ind] + b[ind];
}

int main() {
	int *a, *b, *c;
	hipMallocManaged(&a, M*N*sizeof(int));
	hipMallocManaged(&b, M*N*sizeof(int));
	hipMallocManaged(&c, M*N*sizeof(int));

	for (int i = 0; i < M * N; i++)
	{
		a[i] = i;
		b[i] = 2 * i;
	}

	dim3 blocks(N / thx, M / thy);
	dim3 threads(thx, thy);

	add<<< blocks, threads >>>(a, b, c);
	hipDeviceSynchronize();

	for (int i = 0; i < N; i++) {
		for (int j = 0; j < M; j++)
			printf("%d ", c[i*N + j]);
		printf("\n");
	}

	hipFree(a);
	hipFree(b);
	hipFree(c);
}
